#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <chrono>
#include <math.h>
#include <hip/hip_runtime.h>
#include "reference.h"

template <typename T, typename G>
__global__
void adam (
        T* __restrict__ p,
        T* __restrict__ m,
        T* __restrict__ v,
  const G* __restrict__ g,
  const float b1,
  const float b2,
  const float eps,
  const float grad_scale,
  const float step_size,
  const size_t tsize,
  adamMode_t mode,
  const float decay)
{
  const int i = blockIdx.x * blockDim.x + threadIdx.x;
  const int totThreads = gridDim.x*blockDim.x;

  for (size_t j = i; j < tsize; j += totThreads) {
    T scaled_grad = g[j]/grad_scale;
    m[j] = b1*m[j] + (1.f-b1)*scaled_grad;
    v[j] = b2*v[j] + (1.f-b2)*scaled_grad*scaled_grad;
    float denom;
    if (mode == ADAM_MODE_0)
      denom = sqrtf(v[j] + eps);
    else // Mode 1
      denom = sqrtf(v[j]) + eps;
    float update = (m[j]/denom) + (decay*p[j]);
    p[j] -= (step_size*update);
  }
}

int main(int argc, char* argv[])
{
  if (argc != 3) {
    printf("Usage: %s <size> <repeat>\n", argv[0]);
    return 1;
  }

  const int tsize = atoi(argv[1]);
  const int repeat = atoi(argv[2]);

  size_t size_bytes = tsize * sizeof(float);

  float *m = (float*) malloc (size_bytes);
  float *v = (float*) malloc (size_bytes);
  float *g = (float*) malloc (size_bytes);
  float *p = (float*) malloc (size_bytes);
  float *r = (float*) malloc (size_bytes);

  srand(123);
  for (int i = 0; i < tsize; i++) {
    m[i] = rand() / (float)RAND_MAX;
    v[i] = rand() / (float)RAND_MAX;
    g[i] = rand() / (float)RAND_MAX;
    r[i] = p[i] = rand() / (float)RAND_MAX;
  }

  float *d_m, *d_v, *d_g, *d_p;

  hipMalloc((void**)&d_m, size_bytes);
  hipMemcpy(d_m, m, size_bytes, hipMemcpyHostToDevice);

  hipMalloc((void**)&d_v, size_bytes);
  hipMemcpy(d_v, v, size_bytes, hipMemcpyHostToDevice);

  hipMalloc((void**)&d_g, size_bytes);
  hipMemcpy(d_g, g, size_bytes, hipMemcpyHostToDevice);

  hipMalloc((void**)&d_p, size_bytes);
  hipMemcpy(d_p, p, size_bytes, hipMemcpyHostToDevice);

  // Arbitrary constants
  const float step_size = 1e-3f;
  const float decay = 0.5f;
  const float beta1 = 0.9f;
  const float beta2 = 0.999f;
  const float eps = 1e-10f;
  const float grad_scale = 256.f;

  const int threadsPerBlock = 256;
  const dim3 grids ((tsize+threadsPerBlock-1) / threadsPerBlock);
  const dim3 blocks (threadsPerBlock);

  adamMode_t mode = ADAM_MODE_0;

  hipDeviceSynchronize();
  auto start = std::chrono::steady_clock::now();

  for (int i = 0; i < repeat; i++) {
    adam<float, float><<<grids, blocks>>> (
      d_p, d_m, d_v, d_g,
      beta1, beta2,
      eps,
      grad_scale,
      step_size,
      tsize,
      mode,
      decay);
  }

  hipDeviceSynchronize();
  auto end = std::chrono::steady_clock::now();
  auto time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
  printf("Average kernel execution time %f (ms)\n", time * 1e-6f / repeat);

  hipMemcpy(p, d_p, size_bytes, hipMemcpyDeviceToHost); 

  hipFree(d_p);
  hipFree(d_m);
  hipFree(d_v);
  hipFree(d_g);

  // verify
  reference<float, float>(
    repeat,
    r, m, v, g,
    beta1, beta2,
    eps,
    grad_scale,
    step_size,
    tsize,
    mode,
    decay);

  bool ok = true; 
  for (int i = 0; i < tsize; i++) {
    if (r[i] - p[i] > 1e-3f) {
      ok = false;
      break;
    }
  }
  printf("%s\n", ok ? "PASS" : "FAIL");

  free(p);
  free(m);
  free(v);
  free(g);
  free(r);
  return 0;
}
