
#include <hip/hip_runtime.h>
void ccsd_tengy_gpu(const float * __restrict__ f1n,    const float * __restrict__ f1t,
                    const float * __restrict__ f2n,    const float * __restrict__ f2t,
                    const float * __restrict__ f3n,    const float * __restrict__ f3t,
                    const float * __restrict__ f4n,    const float * __restrict__ f4t,
                    const float * __restrict__ dintc1, const float * __restrict__ dintx1, const float * __restrict__ t1v1,
                    const float * __restrict__ dintc2, const float * __restrict__ dintx2, const float * __restrict__ t1v2,
                    const float * __restrict__ eorb,   const float eaijk,
                    float * __restrict__ emp4i, float * __restrict__ emp5i,
                    float * __restrict__ emp4k, float * __restrict__ emp5k,
                    const int ncor, const int nocc, const int nvir);

void ccsd_trpdrv(float * __restrict__ f1n, float * __restrict__ f1t,
                 float * __restrict__ f2n, float * __restrict__ f2t,
                 float * __restrict__ f3n, float * __restrict__ f3t,
                 float * __restrict__ f4n, float * __restrict__ f4t,
                 float * __restrict__ eorb,
                 int    * __restrict__ ncor_, int * __restrict__ nocc_, int * __restrict__ nvir_,
                 float * __restrict__ emp4_, float * __restrict__ emp5_,
                 int    * __restrict__ a_, int * __restrict__ i_, int * __restrict__ j_, int * __restrict__ k_, int * __restrict__ klo_,
                 float * __restrict__ tij, float * __restrict__ tkj, float * __restrict__ tia, float * __restrict__ tka,
                 float * __restrict__ xia, float * __restrict__ xka, float * __restrict__ jia, float * __restrict__ jka,
                 float * __restrict__ kia, float * __restrict__ kka, float * __restrict__ jij, float * __restrict__ jkj,
                 float * __restrict__ kij, float * __restrict__ kkj,
                 float * __restrict__ dintc1, float * __restrict__ dintx1, float * __restrict__ t1v1,
                 float * __restrict__ dintc2, float * __restrict__ dintx2, float * __restrict__ t1v2)
{
    float emp4 = *emp4_;
    float emp5 = *emp5_;

    float emp4i = 0.0;
    float emp5i = 0.0;
    float emp4k = 0.0;
    float emp5k = 0.0;

    const int ncor = *ncor_;
    const int nocc = *nocc_;
    const int nvir = *nvir_;

    /* convert from Fortran to C offset convention... */
    const int k   = *k_ - 1;
    //const int klo = *klo_ - 1;
    const int a   = *a_ - 1;
    const int i   = *i_ - 1;
    const int j   = *j_ - 1;

    const float eaijk = eorb[a] - (eorb[ncor+i] + eorb[ncor+j] + eorb[ncor+k]);

    ccsd_tengy_gpu(f1n, f1t, f2n, f2t, f3n, f3t, f4n, f4t,
                   dintc1, dintx1, t1v1, dintc2, dintx2, t1v2,
                   eorb, eaijk, &emp4i, &emp5i, &emp4k, &emp5k,
                   ncor, nocc, nvir);

    emp4 += emp4i;
    emp5 += emp5i;

    if (*i_ != *k_) {
        emp4 += emp4k;
        emp5 += emp5k;
    }

    *emp4_ = emp4;
    *emp5_ = emp5;

    return;
}
