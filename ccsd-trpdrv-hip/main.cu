
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

/* Do not allow the test to allocate more than MAX_MEM gigabytes. */
#ifndef MAX_MEM
#define MAX_MEM 4
#endif

#define MIN(x,y) (x<y ? x : y)
#define MAX(x,y) (x>y ? x : y)

void ccsd_trpdrv(
    float * __restrict__ f1n, float * __restrict__ f1t,
    float * __restrict__ f2n, float * __restrict__ f2t,
    float * __restrict__ f3n, float * __restrict__ f3t,
    float * __restrict__ f4n, float * __restrict__ f4t,
    float * __restrict__ eorb,
    int    * __restrict__ ncor_, int * __restrict__ nocc_, int * __restrict__ nvir_,
    float * __restrict__ emp4_, float * __restrict__ emp5_,
    int    * __restrict__ a_, int * __restrict__ i_, int * __restrict__ j_, int * __restrict__ k_, int * __restrict__ klo_,
    float * __restrict__ tij, float * __restrict__ tkj, float * __restrict__ tia, float * __restrict__ tka,
    float * __restrict__ xia, float * __restrict__ xka, float * __restrict__ jia, float * __restrict__ jka,
    float * __restrict__ kia, float * __restrict__ kka, float * __restrict__ jij, float * __restrict__ jkj,
    float * __restrict__ kij, float * __restrict__ kkj,
    float * __restrict__ dintc1, float * __restrict__ dintx1, float * __restrict__ t1v1,
    float * __restrict__ dintc2, float * __restrict__ dintx2, float * __restrict__ t1v2);

float * make_array(int n)
{
  float * a = (float*) malloc(n*sizeof(float));
  for (int i=0; i<n; i++) {
    a[i] = drand48();
  }
  return a;
}

int main(int argc, char* argv[])
{
  int ncor, nocc, nvir;
  int maxiter = 100;
  int nkpass = 1;

  if (argc<3) {
    printf("Usage: ./test_cbody nocc nvir [maxiter] [nkpass]\n");
    return argc;
  } else {
    ncor = 0;
    nocc = atoi(argv[1]);
    nvir = atoi(argv[2]);
    if (argc>3) {
      maxiter = atoi(argv[3]);
      /* if negative, treat as "infinite" */
      if (maxiter<0) maxiter = 1<<30;
    }
    if (argc>4) {
      nkpass = atoi(argv[4]);
    }
  }

  if (nocc<1 || nvir<1) {
    printf("Arguments must be non-negative!\n");
    return 1;
  }

  printf("Test driver for cbody with nocc=%d, nvir=%d, maxiter=%d, nkpass=%d\n", nocc, nvir, maxiter, nkpass);

  const int nbf = ncor + nocc + nvir;
  const int lnvv = nvir * nvir;
  const int lnov = nocc * nvir;
  const int kchunk = (nocc - 1)/nkpass + 1;

  const float memory = (nbf+8.0*lnvv+
      lnvv+kchunk*lnvv+lnov*nocc+kchunk*lnov+lnov*nocc+kchunk*lnov+lnvv+
      kchunk*lnvv+lnvv+kchunk*lnvv+lnov*nocc+kchunk*lnov+lnov*nocc+
      kchunk*lnov+lnov+nvir*kchunk+nvir*nocc+
      6.0*lnvv)*sizeof(float);
  printf("This test requires %f GB of memory.\n", 1.0e-9*memory);

  if (1.0e-9*memory > MAX_MEM) {
    printf("You need to increase MAX_MEM (%d)\n", MAX_MEM);
    printf("or set nkpass (%d) to a larger number.\n", nkpass);
    return MAX_MEM;
  }

  srand48(2);
  float * eorb = make_array(nbf);
  float * f1n = make_array(lnvv);
  float * f2n = make_array(lnvv);
  float * f3n = make_array(lnvv);
  float * f4n = make_array(lnvv);
  float * f1t = make_array(lnvv);
  float * f2t = make_array(lnvv);
  float * f3t = make_array(lnvv);
  float * f4t = make_array(lnvv);
  float * Tij = make_array(lnvv);
  float * Tkj = make_array(kchunk*lnvv);
  float * Tia = make_array(lnov*nocc);
  float * Tka = make_array(kchunk*lnov);
  float * Xia = make_array(lnov*nocc);
  float * Xka = make_array(kchunk*lnov);
  float * Jia = make_array(lnvv);
  float * Jka = make_array(kchunk*lnvv);
  float * Kia = make_array(lnvv);
  float * Kka = make_array(kchunk*lnvv);
  float * Jij = make_array(lnov*nocc);
  float * Jkj = make_array(kchunk*lnov);
  float * Kij = make_array(lnov*nocc);
  float * Kkj = make_array(kchunk*lnov);
  float * Dja = make_array(lnov);
  float * Djka = make_array(nvir*kchunk);
  float * Djia = make_array(nvir*nocc);
  float * dintc1 = make_array(lnvv);
  float * dintc2 = make_array(lnvv);
  float * dintx1 = make_array(lnvv);
  float * dintx2 = make_array(lnvv);
  float * t1v1 = make_array(lnvv);
  float * t1v2 = make_array(lnvv);

  int ntimers = MIN(maxiter,nocc*nocc*nocc*nocc);
  float * timers = (float*) calloc(ntimers,sizeof(float));

  float emp4=0.0, emp5=0.0;

  int iter = 0;

  for (int klo=1; klo<=nocc; klo+=kchunk) {
    const int khi = MIN(nocc, klo+kchunk-1);
    int a=1;
    for (int j=1; j<=nocc; j++) {
      for (int i=1; i<=nocc; i++) {
        for (int k=klo; k<=MIN(khi,i); k++) {
          clock_t t0 = clock();
          ccsd_trpdrv(f1n, f1t, f2n, f2t, f3n, f3t, f4n, f4t, eorb,
              &ncor, &nocc, &nvir, &emp4, &emp5, &a, &i, &j, &k, &klo,
              Tij, Tkj, Tia, Tka, Xia, Xka, Jia, Jka, Kia, Kka, Jij, Jkj, Kij, Kkj,
              dintc1, dintx1, t1v1, dintc2, dintx2, t1v2);
          timers[iter] = (float)(clock()-t0) / CLOCKS_PER_SEC;

          iter++;
          if (iter==maxiter) {
            printf("Stopping after %d iterations...\n", iter);
            goto maxed_out;
          }

          /* prevent NAN for large maxiter... */
          if (emp4 >  1000.0) emp4 -= 1000.0;
          if (emp4 < -1000.0) emp4 += 1000.0;
          if (emp5 >  1000.0) emp5 -= 1000.0;
          if (emp5 < -1000.0) emp5 += 1000.0;
        }
      }
    }
  }

maxed_out:
  float tsum =  0.0;
  float tmax = -1.0e10;
  float tmin =  1.0e10;
  for (int i=0; i<iter; i++) {
    tsum += timers[i];
    tmax  = MAX(tmax,timers[i]);
    tmin  = MIN(tmin,timers[i]);
  }
  float tavg = tsum / iter;
  printf("TIMING: min=%lf, max=%lf, avg=%lf\n", tmin, tmax, tavg);

  float dgemm_flops = ((8.0*nvir)*nvir)*(nvir+nocc);
  float dgemm_mops  = 8.0*(4.0*nvir*nvir + 2.0*nvir*nocc);

  /* The inner loop of tengy touches 86 f[1234][nt] elements and 8 other arrays...
   * We will just assume flops=mops even though flops>mops */
  float tengy_ops = ((1.0*nvir)*nvir)*(86+8);

  printf("OPS: dgemm_flops=%10.3e dgemm_mops=%10.3e tengy_ops=%10.3e\n",
         dgemm_flops, dgemm_mops, tengy_ops);

  printf("PERF: GF/s=%10.3e GB/s=%10.3e\n",
         1.0e-9*(dgemm_flops+tengy_ops)/tavg, 8.0e-9*(dgemm_mops+tengy_ops)/tavg);

  printf("These are meaningless but should not vary for a particular input:\n");
  printf("emp4=%f emp5=%f\n", emp4, emp5);
  printf("Finished\n");

  free(eorb);
  free(f1n );
  free(f2n );
  free(f3n );
  free(f4n );
  free(f1t );
  free(f2t );
  free(f3t );
  free(f4t );
  free(Tij );
  free(Tkj );
  free(Tia );
  free(Tka );
  free(Xia );
  free(Xka );
  free(Jia );
  free(Jka );
  free(Kia );
  free(Kka );
  free(Jij );
  free(Jkj );
  free(Kij );
  free(Kkj );
  free(Dja );
  free(Djka);
  free(Djia);
  free(dintc1);
  free(dintc2);
  free(dintx1);
  free(dintx2);
  free(t1v1  );
  free(t1v2  );

  return 0;
}
