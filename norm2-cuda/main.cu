#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

long get_time() {
  struct timeval tv;
  gettimeofday(&tv, NULL);
  return (tv.tv_sec * 1000000) + tv.tv_usec;
}

int main (int argc, char* argv[]){
  if (argc != 2) {
    printf("Usage: %s <repeat>\n", argv[0]);
    return 1;
  }

  // repeat at least once
  const int repeat = max(1, atoi(argv[1]));

  bool ok = true;
  hipError_t cudaStat;
  hipblasStatus_t cublasStat;
  hipblasHandle_t handle;

  cublasStat = hipblasCreate(&handle);
  if (cublasStat != HIPBLAS_STATUS_SUCCESS) {
    printf ("CUBLAS initialization failed\n");
  }

  // store the nrm2 results
  float* result = (float*) malloc (repeat * sizeof(float));
  if (!result) {
    printf ("result memory allocation failed");
    return 1;
  }

  for (int n = 512*1024; n <= 1024*1024*512; n = n * 2) {
    int i, j;
    size_t size = n * sizeof(float);
    float* a = (float *) malloc (size);
    if (!a) {
      printf ("host memory allocation failed");
      break;
    }

    // reference
    double gold = 0.0;  // double is required to match host and device results 
    for (i = 0; i < n; i++) {
      a[i] = (float)((i+1) % 7);
      gold += a[i]*a[i];
    }
    gold = sqrt(gold);

    long start = get_time();

    float* d_a;
    cudaStat = hipMalloc ((void**)&d_a, size);
    if (cudaStat != hipSuccess) {
      printf ("device memory allocation failed");
    }

    cudaStat = hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    if (cudaStat != hipSuccess) {
      printf ("device memory copy failed");
    }

    for (j = 0; j < repeat; j++) {
      cublasStat = hipblasSnrm2(handle, n, d_a, 1, result+j);
      if (cublasStat != HIPBLAS_STATUS_SUCCESS) {
        printf ("CUBLAS Snrm2 failed\n");
      }
    }

    cudaStat = hipFree(d_a);
    if (cudaStat != hipSuccess) {
      printf ("device memory deallocation failed");
    }

    long end = get_time();
    printf("#elements = %.2f M, measured time = %.3f s\n", 
            n / (1024.f*1024.f), (end-start) / 1e6f);

    if (a != NULL) free(a);

    // snrm2 results match across all iterations
    for (j = 0; j < repeat; j++) 
     if (fabsf((float)gold - result[j]) > 1e-3f) {
       printf("FAIL at iteration %d: gold=%f actual=%f for %d elements\n",
              j, (float)gold, result[j], i);
       ok = false;
       break;
     }
  }

  free(result);
  hipblasDestroy(handle);

  if (ok) printf("PASS\n");
  return 0;
}
