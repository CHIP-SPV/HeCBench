#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <chrono>
#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>
#include <thrust/sequence.h>
#include <thrust/sort.h>
#include <thrust/system/hip/execution_policy.h>
#include <thrust/unique.h>
#include <thrust/version.h>

#define NUM_THREADS 256

__global__
void remap_kernel(
  thrust::device_ptr<const int> second_order,
  thrust::device_ptr<const int> first_order,
  int *output,
  const int N,
  const int K)
{
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i >= K) return;
  int idx = second_order[i];
  output[first_order[idx]] = i;
  for (idx++; idx < N && (i == K - 1 || idx != second_order[i + 1]); idx++) {
    output[first_order[idx]] = i;
  }
}

template <typename T>
void eval_remap(const int N, const int repeat) {

  size_t input_size_bytes = N * sizeof(T);
  size_t output_size_bytes = N * sizeof(int);

  int *h_input = (int*) malloc (input_size_bytes);

#ifdef EXAMPLE
  h_input[0] = 1; h_input[1] = 3; h_input[2] = 5;
  h_input[3] = 1; h_input[4] = 5; h_input[5] = 7;
  h_input[6] = 9;
#else
  srand(123);
  for (int i = 0; i < N; i++) {
    h_input[i] = rand() % N;
  }
#endif

  int *h_output = (int*) malloc (output_size_bytes);

  T *d_input;
  hipMalloc((void**)&d_input, input_size_bytes);
  hipMemcpy(d_input, h_input, input_size_bytes, hipMemcpyHostToDevice);

  int *d_output;
  hipMalloc((void**)&d_output, output_size_bytes);

  auto start = std::chrono::steady_clock::now();

  // Create two vectors of {0, 1, ..., N-1} on device
  thrust::device_vector<int> order1(N), order2(N);
  thrust::sequence(order1.begin(), order1.end());
  thrust::sequence(order2.begin(), order2.end());

  auto end = std::chrono::steady_clock::now();
  auto time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
  printf("Execution time of sequence : %f (us)\n", (time * 1e-3f));

  // Sort the input along with order vector. So now we know where each element
  // is permutated to. For example:
  //    input1 = 1,3,5,1,5,7,9
  //    order1 = 0,1,2,3,4,5,6
  // Now we have:
  //    output = 1,1,3,5,5,7,9
  //    order1 = 0,3,1,2,4,5,6
  start = std::chrono::steady_clock::now();

  auto buffer = thrust::device_pointer_cast(d_input);
  thrust::sort_by_key(buffer, buffer + N, order1.begin());

  end = std::chrono::steady_clock::now();
  time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
  printf("Execution time of sort-by-key : %f (us)\n", (time * 1e-3f));

  // Use consequent unique op to get another order_buffer
  //    input2 = 1,1,3,5,5,7,9
  //    order2 = 0,1,2,3,4,5,6
  // Now we have:
  //    output = 1,3,5,7,9
  //    order2 = 0,2,3,5,6
  start = std::chrono::steady_clock::now();

  auto result = thrust::unique_by_key(buffer, buffer + N, order2.begin());

  end = std::chrono::steady_clock::now();
  time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
  printf("Execution time of unique-by-key : %f (us)\n", (time * 1e-3f));

  int K = result.first - buffer;
  printf("Percentage of unique elements: %.1f %%\n", (float) K * 100 / N);

  // Compute the remapping. For example, for the number 1, if we look at
  // order2[0] and order2[1], we know that input2[0:2) are all 1. They are all
  // remapped to 0 in final input. And from order1, we know where they come from.
  dim3 grid ((K + NUM_THREADS - 1) / NUM_THREADS);
  dim3 block (NUM_THREADS);

  hipDeviceSynchronize();
  start = std::chrono::steady_clock::now();

  for (int i = 0; i < repeat; i++)
    remap_kernel<<<grid, block>>>(order2.data(), order1.data(), d_output, N, K);

  hipDeviceSynchronize();
  end = std::chrono::steady_clock::now();
  time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
  printf("Average execution time of remap kernel: %f (us)\n",
         (time * 1e-3f) / repeat);

  hipMemcpy(h_output, d_output, output_size_bytes, hipMemcpyDeviceToHost);

  int cs1 = 0, cs2 = 0;
  for (int i = 0; i < N-1; i++) {
    cs1 ^= h_output[i] - h_output[i+1];
  }
  for (int i = 0; i < N; i++) {
    cs2 ^= h_output[i];
  }
  printf("Checksum: %d %d\n", cs1, cs2);

#ifdef EXAMPLE
  for (int i = 0; i < N; i++) {
    printf("%d ", h_output[i]);
  }
  printf("\n");
#endif

  hipFree(d_output);
  hipFree(d_input);

  free(h_output);
  free(h_input);
}

int main(int argc, char* argv[])
{
  if (argc != 3) {
    printf("Usage: %s <number of elements> <repeat>\n", argv[0]);
    return 1;
  }

#ifdef EXAMPLE
  const int N = 7;
#else
  const int N = atoi(argv[1]);
#endif
  const int repeat = atoi(argv[2]);

  // warmup and run 
  for (int i = 0; i < 4; i++) {
    printf("\nRun %d\n", i);
    eval_remap<int>(N, repeat);
  }

  return 0;
}
